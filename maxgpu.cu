#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

unsigned int getmaxcu(unsigned int *, unsigned int);

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array

    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *) malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++){
       numbers[i] = rand() % size;
       printf("%d\n", numbers[i]);
    }
    unsigned int max = getmaxcu(numbers, size);
    printf(" The maximum number in the array is: %u\n", max);
    free(numbers);
    exit(0);
}//end of main


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/

__global__ void getmaxcu(unsigned int* globalInputArr, unsigned int* globalOutputArr, unsigned int* sizeArr){
	//you need a shared array (per block) to put the block's max into
	//you also need a global array to put the overall max into
	extern __shared__ unsigned int sdata[];

	unsigned int size = sizeArr[0];

	unsigned int tid = threadIdx.x; 
	unsigned int gid = (blockIdx.x * blockDim.x) + threadIdx.x; //getting the unique index of thread
	sdata[tid] = 0; //initializing the shared data array (shared per block)
	
	if(gid < size){
		sdata[tid] = globalInputArr[gid];
	}
	__syncthreads();
	/*
	for (unsigned int s = blockDim.x/2; s>0; s>>=1) {
		if(gid < size && tid < s) {
			sdata[tid] = max(sdata[tid], sdata[tid + s]);
		}
		__syncthreads();
	}
	*/
	if (tid == 0){
		globalOutputArr[blockIdx.x] = sdata[tid]; //putting all the max from each block into a global output array
	}
}

__global__ void finalmaxcu(unsigned int* globalOutputArr, unsigned int* max){
	int tid = threadIdx.x;
	extern __shared__ unsigned int sdata[];
	sdata[tid] = 0;

	if(tid < blockDim.x){
		sdata[tid] = globalOutputArr[tid];
	}
	__syncthreads();
	/*
	for (unsigned int s=blockDim.x/2; s>0; s>>=1){ //it starts at the half way mark and keeps div in 2
		if(tid < s){
			unsigned int greater = sdata[tid];
			if(sdata[tid] < sdata[tid+s]){
				greater = sdata[tid+s];
			}
			sdata[tid] = greater;
		}
		__syncthreads();
	}
	*/
	if (tid == 0){
		max[0] = sdata[tid];
	}
}

unsigned int getmaxcu(unsigned int* numbers, unsigned int num_elem){
	//max num of threads per SM : 2048
	//max num of threads per block : 1024

	unsigned int* sizeArr = (unsigned int*) malloc(sizeof(unsigned int));//creating an array to pass on to the device
	sizeArr[0] = num_elem;
	unsigned int* size; //declaring a size integer (device)
	hipMalloc((void**)&size, sizeof(unsigned int)); 
	hipMemcpy((void*) size, (void*) sizeArr, sizeof(unsigned int), hipMemcpyHostToDevice);
	
	unsigned int* globalInputArr;
	hipMalloc((void**)&globalInputArr, num_elem * sizeof(unsigned int));
	hipMemcpy((void*) globalInputArr, (void*) numbers, num_elem * sizeof(unsigned int), hipMemcpyHostToDevice);
	
	unsigned int* globalOutputArr;
	hipMalloc((void**)&globalOutputArr, num_elem*sizeof(unsigned int));

	unsigned int* max;
	hipMalloc((void**)&max, sizeof(unsigned int));//allocating the max number pointer in the device

	unsigned int* maxNum = (unsigned int*) malloc(sizeof(unsigned int)); //allocating the max number pointer in the host
	
	//first experimenting with block size of 128, the max block size is 1024
	//adding size as the third parameter in the triple bracket sets the byte size for the sdata (which is in the shared memory)
	//whatever is in sdata should be the size of N divided by the number of blocks (which is 8 for now)
	unsigned int sharedSize = (num_elem / 8) * sizeof(unsigned int);
	getmaxcu<<<8, 128, sharedSize>>>(globalInputArr, globalOutputArr, size); 

	unsigned int* copy = (unsigned int*) malloc(8 * sizeof(unsigned int));
	hipMemcpy((void*) copy, (void*) globalOutputArr, (num_elem * sizeof(unsigned int)), hipMemcpyDeviceToHost);
	for(int i = 0; i < 8; i++){
		printf("%u,",copy[i]);
	}

	printf("\n");
	finalmaxcu<<<1, 128, (8 * sizeof(unsigned int))>>>(globalOutputArr, max);
	hipMemcpy((void*) maxNum, (void*) max, sizeof(unsigned int), hipMemcpyDeviceToHost);//copying back the max from the device to host
   	
    hipFree(max);
	hipFree(globalInputArr);
	hipFree(globalOutputArr);
	return maxNum[0];
}




unsigned int getmax(unsigned int num[], unsigned int size)
{

  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );

}